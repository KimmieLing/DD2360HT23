//OPTIMIZE AND CHECK SO I USE SHARED MEMORY PROPERLY



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <random>

#define NUM_BINS 4096

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

//@@ Insert code below to compute histogram of input using shared memory and atomics
__shared__ unsigned int localBins[NUM_BINS];

int index = blockIdx.x * blockDim.x + threadIdx.x;

// Initialize localBins in shared memory to zeros
    if (threadIdx.x < num_bins) {
        localBins[threadIdx.x] = 0;
    }

    __syncthreads();

//Add counter for localbin that corresponds with input value
if(index < num_elements)
{
  atomicAdd(&localBins[threadIdx.x], 1);
}

__syncthreads();

if(threadIdx.x < num_bins)
{
  atomicAdd(&bins[input[index]], localBins[threadIdx.x]);
}

}

__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {

//@@ Insert code below to clean up bins that saturate at 127
int index = blockIdx.x * blockDim.x + threadIdx.x;

if(index < num_bins && bins[index] > 127)
{
  bins[index] = 127;
}

}


int main(int argc, char **argv) {
  
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *resultRef;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  //@@ Insert code below to read in inputLength from args
  inputLength = std::atoi(argv[1]);

  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hipHostMalloc(&hostInput, inputLength*sizeof(unsigned int), hipHostMallocDefault);
  hipHostMalloc(&hostBins, NUM_BINS*sizeof(unsigned int), hipHostMallocDefault);
  hipHostMalloc(&resultRef, inputLength*sizeof(unsigned int), hipHostMallocDefault);

  
  //@@ Insert code below to initialize hostInput to random numbers whose values range from 0 to (NUM_BINS - 1)
  for(int i = 0; i < inputLength; i++)
  {
    hostInput[i] = rand() % NUM_BINS;
  }


  //@@ Insert code below to create reference result in CPU
  //initialize result to zero
  for(int i = 0; i < NUM_BINS; i++)
  {
    resultRef[i] = 0;
  }

  //Histogram, and also saturize if resultref is 127
  for(int i = 0; i < inputLength; i++)
  {
    if(resultRef[hostInput[i]] < 127)
    {
      resultRef[hostInput[i]] += 1;
    }
  }


  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput,inputLength*sizeof(unsigned int));
  hipMalloc(&deviceBins,NUM_BINS*sizeof(unsigned int));


  //@@ Insert code to Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, inputLength * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(deviceBins, hostBins, NUM_BINS*sizeof(unsigned int), hipMemcpyHostToDevice);

  //@@ Insert code to initialize GPU results
  hipMemset(deviceBins, 0, inputLength*sizeof(unsigned int));

  //@@ Initialize the grid and block dimensions here
  //For the first kernel we want to launch the same amount of threads as inputLength:
  int blockSize(256);
  int gridSize((inputLength+blockSize -1)/blockSize);

  //@@ Launch the GPU Kernel here
  histogram_kernel<<<dim3(gridSize), dim3(blockSize)>>>(deviceInput, deviceBins, inputLength, NUM_BINS);
  hipDeviceSynchronize();

  //@@ Initialize the second grid and block dimensions here
  //For second kernel we want to launch one kernel for each bin:
  int convertBlockSize(256);
  int convertGridSize((NUM_BINS+convertBlockSize -1)/convertBlockSize);


  //@@ Launch the second GPU Kernel here
  convert_kernel<<<dim3(convertGridSize), dim3(convertBlockSize)>>>(deviceBins, NUM_BINS);
  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins, deviceBins, NUM_BINS*sizeof(unsigned int), hipMemcpyDeviceToHost);


  //@@ Insert code below to compare the output with the reference
  int resultCounter = 0;
  for(int i = 0; i < NUM_BINS; i++)
  {
    if(resultRef[i] == hostBins[i])
    {
      resultCounter++;
    }else{
      printf("Reference Bin: %d\n Device Bin: %d\n index: %d\n", resultRef[i], hostBins[i], i);
    }
  }
  printf("Amount correct: %d/%d", resultCounter, NUM_BINS);

  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);


  //@@ Free the CPU memory here
  hipHostFree(hostInput);
  hipHostFree(hostBins);
  hipHostFree(resultRef);

  return 0;
}

